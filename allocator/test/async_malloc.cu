#include "hip/hip_runtime.h"
#include <util.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <glog/logging.h>

#define CUDA_CALL(func) do { \
    auto error = func; \
    if (error != hipSuccess) { \
        LOG(FATAL) << #func << " " << hipGetErrorString(error); \
        exit(EXIT_FAILURE); \
    } \
    } while (0)

#define CU_CALL(func) \
    do { \
        auto err = func; \
        if (err != hipSuccess) { \
            const char* pstr = nullptr; \
            hipDrvGetErrorString(err, &pstr); \
            LOG(FATAL) << #func << ": " << pstr; \
            exit(EXIT_FAILURE); \
        } \
    } while (0);

using namespace mpool;

__global__ void kernel() {
    while(true) {}
}

int main1() {
    CU_CALL(hipInit(0));

    hipCtx_t cuContext;
    CU_CALL(hipCtxCreate(&cuContext, hipDeviceScheduleBlockingSync, 0));
    void *dev_ptr;
    // CUDA_CALL(hipMallocAsync(&dev_ptr, 32_MB));
    CU_CALL(hipMemAddressReserve(reinterpret_cast<hipDeviceptr_t *>(&dev_ptr), 32_MB, 32_MB, 0, 0));
    hipMemGenericAllocationHandle_t cu_handle;
    hipMemAllocationProp prop = {
        .type = hipMemAllocationTypePinned,
        .requestedHandleTypes = hipMemHandleTypePosixFileDescriptor,
        .location = {
        .type = hipMemLocationTypeDevice,
        .id = 0
        }
    };
    CU_CALL(hipMemCreate(&cu_handle, 32_MB, &prop, 0));
    CU_CALL(hipMemMap( reinterpret_cast<hipDeviceptr_t>(dev_ptr), 32_MB, 0, cu_handle, 0));
    hipMemAccessDesc acc_desc = {
        .location = {.type = hipMemLocationTypeDevice, .id = 0},
        .flags = hipMemAccessFlagsProtReadWrite
    };
    CU_CALL(hipMemSetAccess(reinterpret_cast<hipDeviceptr_t>(dev_ptr), 32_MB, &acc_desc, 1));
    hipStream_t cu_stream;
    CU_CALL(hipStreamCreateWithFlags(&cu_stream, hipStreamDefault));
    hipEvent_t cu_event;
    CU_CALL(hipEventCreateWithFlags(&cu_event, hipEventBlockingSync));
    kernel<<<1,1,0, cu_stream>>>();
    void *dev_ptr1;
    CUDA_CALL(hipMallocAsync(&dev_ptr1, 32_MB, cu_stream));
    CU_CALL(hipEventRecord(cu_event, cu_stream));
    // CU_CALL(hipEventSynchronize(cu_event));
    CUDA_CALL(hipMemsetAsync(dev_ptr, 0, 32_MB, cu_stream));
    CU_CALL(hipStreamWaitEvent(cu_stream, cu_event, 0));
    CU_CALL(hipStreamSynchronize(cu_stream));
    CUDA_CALL(hipStreamSynchronize(cu_stream));

    
    hipStream_t cu_stream1;
    CU_CALL(hipStreamCreateWithFlags(&cu_stream1, hipStreamDefault));
    CU_CALL(hipStreamWaitEvent(cu_stream1, cu_event, CU_EVENT_WAIT_DEFAULT));
    CUDA_CALL(hipMemsetAsync(dev_ptr, 0, 32_MB, cu_stream1));
    CU_CALL(hipStreamSynchronize(cu_stream1));
    LOG(INFO) << "finish";
    return 0;
}

int main() {
    CU_CALL(hipInit(0));

    hipCtx_t cuContext;
    CU_CALL(hipCtxCreate(&cuContext, hipDeviceScheduleBlockingSync, 0));

    hipStream_t cu_stream;
    CU_CALL(hipStreamCreateWithFlags(&cu_stream, hipStreamDefault));

    // void *flag = 0;
    // CU_CALL(hipMemAllocHost(&flag, sizeof(cuuint64_t)));
    hipDeviceptr_t flag_dev;
    CUDA_CALL(hipMalloc(reinterpret_cast<void **>(&flag_dev), sizeof(cuuint64_t)));
    CU_CALL(hipStreamWriteValue32(cu_stream, flag_dev, 0, CU_STREAM_WRITE_VALUE_DEFAULT ));
    CU_CALL(hipStreamWaitValue32(cu_stream, flag_dev, 1, hipStreamWaitValueEq ));

    CUDA_CALL(hipMemsetAsync(0, 0, 32_MB, cu_stream));
}